#include "hip/hip_runtime.h"
/**
 *   Alexandre Pinto e Eduardo Fernandes, Maio 2023
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common.h"
#include <hip/hip_runtime.h>

/**
 *   program configuration
 */

// definir N
// shiftar se for preciso

#ifndef SECTOR_SIZE
# define SECTOR_SIZE  512
#endif
#ifndef N_SECTORS
# define N_SECTORS    (1 << 21)                            // it can go as high as (1 << 21)
#endif

/* allusion to internal functions */

//alusão a 2 nucleos computacionais
// ponteiro para o 1 elemento da sequencia, de N

static void modify_sector_cpu_kernel (unsigned int *sector_data, unsigned int sector_number, unsigned int n_sectors,
                                      unsigned int sector_size);
__global__ static void modify_sector_cuda_kernel (unsigned int * __restrict__ sector_data, unsigned int * __restrict__ sector_number,
                                                  unsigned int n_sectors, unsigned int sector_size);
static double get_delta_time(void);

/** @brief Number of integers */
static int num_integers;

/** @brief Name of the file  */
static char *fileName;

/** @brief Array of integers  */
static int* integersArray;

/**
 *   main program
 */

int main (int argc, char **argv)
{
  /*NEW*/

  FILE* file = NULL; // file
  int res; // response from file
  
  /*NEW end*/

  printf("%s Starting...\n", argv[0]);
  if (sizeof (unsigned int) != (size_t) 4)
     return 1;                                             // it fails with prejudice if an integer does not have 4 bytes


  /*NEW*/

  // Checking the number of arguments
  if (argc<2){
    printf("Command is not recognized. Don't forget to enter file name!\n");
    exit(1);
  }

  // Setting the file name
  fileName = argv[1];
  printf("FILE NAME - %s\n",fileName);

  /*NEW end*/

  /* set up the device */
  int dev = 0;
  
  hipDeviceProp_t deviceProp;
  CHECK (hipGetDeviceProperties (&deviceProp, dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  CHECK (hipSetDevice (dev));
  
  /*NEW*/

  // Open file and get number of integers
  file = fopen(fileName, "rb");
  if (file == NULL) {
      fprintf(stderr, "Error opening file %s\n", fileName);
      exit(1);
  }
  res = fread(&num_integers, sizeof(int), 1, file);
  if (res != 1) {
    if (ferror(file)) {
        fprintf(stderr, "Invalid file format\n");
        exit(1);
    }
    else if (feof(file)) {
        printf("Error: end of file reached\n");
        exit(1);
    }
  }

  // Allocate memory for the array of integers in CPU and GPU
  unsigned int *integerArrayGPU;
  integersArray = malloc(num_integers * sizeof(int));
  CHECK (hipMalloc((void**)&integerArrayGPU, num_integers * sizeof(int)));

  /*NEW end*/


  /* create memory areas in host and device memory where the disk sectors data and sector numbers will be stored */

  size_t sector_data_size;
  size_t sector_number_size;
  unsigned int *host_sector_data, *host_sector_number;
  unsigned int *device_sector_data, *device_sector_number;

  sector_data_size = (size_t) N_SECTORS * (size_t) SECTOR_SIZE;
  sector_number_size = (size_t) N_SECTORS * sizeof (unsigned int);
  if ((sector_data_size + sector_number_size) > (size_t) 5e9)
     { fprintf (stderr,"The GeForce GTX 1660 Ti cannot handle more than 5GB of memory!\n");
       exit (1);
     }
  printf ("Total sector data size: %lu\n", sector_data_size);
  printf ("Total sector numbers data size: %lu\n", sector_number_size);

  host_sector_data = (unsigned int *) malloc (sector_data_size);
  host_sector_number = (unsigned int *) malloc (sector_number_size);
  CHECK (hipMalloc ((void **) &device_sector_data, sector_data_size));
  CHECK (hipMalloc ((void **) &device_sector_number, sector_number_size));

  /* initialize the host data */

  int i;

  (void) get_delta_time ();
  srand(0xCCE2021);
  for (i = 0; i < (int) (sector_data_size / (int) sizeof(unsigned int)); i++)
    host_sector_data[i] = 108584447u * (unsigned int) i; // "pseudo-random" data (faster than using the rand() function)
  for(i = 0; i < (int) (sector_number_size / (int)sizeof(unsigned int)); i++)
    host_sector_number[i] = (rand () & 0xFFFF) | ((rand () & 0xFFFF) << 16);
  printf ("The initialization of host data took %.3e seconds\n", get_delta_time ());

  /* copy the host data to the device memory */

  //copy da memoria do cpu para gpu

  // switch até case 9 para definir geometria

  (void) get_delta_time ();
  CHECK (hipMemcpy (device_sector_data, host_sector_data, sector_data_size, hipMemcpyHostToDevice));
  CHECK (hipMemcpy (device_sector_number, host_sector_number, sector_number_size, hipMemcpyHostToDevice));
  printf ("The transfer of %ld bytes from the host to the device took %.3e seconds\n",
          (long) sector_data_size + (long) sector_number_size, get_delta_time ());

  /* run the computational kernel
     as an example, N_SECTORS threads are launched where each thread deals with one sector */

  unsigned int gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ;
  int n_sectors, sector_size;

  n_sectors = N_SECTORS;
  sector_size = SECTOR_SIZE;
  blockDimX = 1 << 0;                                      // optimize!
  blockDimY = 1 << 0;                                      // optimize!
  blockDimZ = 1 << 0;                                      // do not change!
  gridDimX = 1 << 21;                                      // optimize!
  gridDimY = 1 << 0;                                       // optimize!
  gridDimZ = 1 << 0;                                       // do not change!

  dim3 grid (gridDimX, gridDimY, gridDimZ);
  dim3 block (blockDimX, blockDimY, blockDimZ);

  if ((gridDimX * gridDimY * gridDimZ * blockDimX * blockDimY * blockDimZ) != n_sectors)
     { printf ("Wrong configuration!\n");
       return 1;
     }
  (void) get_delta_time ();
  modify_sector_cuda_kernel <<<grid, block>>> (device_sector_data, device_sector_number, n_sectors, sector_size);
  CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish
  CHECK (hipGetLastError ());                                 // check for kernel errors
  printf("The CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3e seconds to run\n",
         gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, get_delta_time ());

  /* copy kernel result back to host side */

  unsigned int *modified_device_sector_data;

  modified_device_sector_data = (unsigned int *) malloc (sector_data_size);
  CHECK (hipMemcpy (modified_device_sector_data, device_sector_data, sector_data_size, hipMemcpyDeviceToHost));
  printf ("The transfer of %ld bytes from the device to the host took %.3e seconds\n",
          (long) sector_data_size, get_delta_time ());

  
  /*NEW*/

  // Deallocate GPU memory
  CHECK (hipFree (integerArrayGPU))
  
  /*NEW end*/
  

  /* free device global memory */

  CHECK (hipFree (device_sector_data));
  CHECK (hipFree (device_sector_number));

  /* reset the device */

  CHECK (hipDeviceReset ());

  /* compute the modified sector data on the CPU */

  (void) get_delta_time ();
  for (i = 0; i < N_SECTORS; i++)
    modify_sector_cpu_kernel (&host_sector_data[i*SECTOR_SIZE/(sizeof (unsigned int))], host_sector_number[i], n_sectors, sector_size);
  printf("The cpu kernel took %.3e seconds to run (single core)\n",get_delta_time ());

  /* compare results */

  for(i = 0; i < (int) sector_data_size / (int) sizeof (unsigned int); i++)
    if (host_sector_data[i] != modified_device_sector_data[i])
       { int sector_words = sector_size / (int) sizeof (unsigned int);

         printf ("Mismatch in sector %d, word %d\n", i / sector_words, i % sector_words);
         exit(1);
       }
  printf ("All is well!\n");


  /*NEW*/

  verifyResults();

  // Deallocate memory
  free(integersArray);
  
  /*NEW end*/
  
  
  /* free host memory */

  free (host_sector_data);
  free (host_sector_number);
  free (modified_device_sector_data);

  return 0;
}

/*
Function to verify if the integers in the final array are sorted correctly
*/
bool verifyResults(){
    printf("\n Final Verification\n");

    int i;
    for (i = 0; i < num_integers - 1; i++)
        if (integersArray[i] > integersArray[i + 1])
        {   
            printf("  Error on file %s!\n", fileName);
            printf("  Error in position %d between element %d and %d\n",
                i, integersArray[i], integersArray[i + 1]);
            return false;
        }
    if (i == (num_integers - 1))
        printf(" Everything is OK for file %s\n", fileName);
    return true;
}


/*
Function to merge
*/
void merge(int* arr, int l, int m, int r) {
    int i, j, k;

    // size of first array
    int n1 = m - l + 1;

    // size of second array
    int n2 = r - m;

    // create temporary arrays you the size of the arrays
    //int L[n1], R[n2];
    int *L = malloc(n1 * sizeof(int));
    int *R = malloc(n2 * sizeof(int));

    // values are copied into the arrays
    for (i = 0; i < n1; i++)
        L[i] = arr[l + i];
    for (j = 0; j < n2; j++)
        R[j] = arr[m + 1 + j];

    i = 0;
    j = 0;
    k = l;

    // iterates through the two subarrays, comparing the values at each index and inserting them into the correct position in the final sorted array arr
    while (i < n1 && j < n2) {
        if (L[i] <= R[j]) {
            arr[k] = L[i];
            i++;
        }
        else {
            arr[k] = R[j];
            j++;
        }
        k++;
    }

    // any remaining elements in L or R are copied over to the final sorted array
    while (i < n1) {
        arr[k] = L[i];
        i++;
        k++;
    }

    while (j < n2) {
        arr[k] = R[j];
        j++;
        k++;
    }

    free(R);
    free(L);
}



/*
Function to merge sort 
*/
void mergeSort(int* arr, int n) {
    int curr_size;
    int left_start;

    // Merge subarrays in bottom-up manner
    for (curr_size = 1; curr_size <= n-1; curr_size = 2*curr_size) {
        // Pick starting point of different subarrays of current size
        for (left_start = 0; left_start < n-1; left_start += 2*curr_size) {
            // Find ending point of left subarray
            int mid = left_start + curr_size - 1;

            // Find ending point of right subarray
            int right_end = MIN(left_start + 2*curr_size - 1, n-1);

            // Merge subarrays arr[left_start...mid] and arr[mid+1...right_end]
            merge(arr, left_start, mid, right_end);
        }
    }
}

static void modify_sector_cpu_kernel (unsigned int *sector_data, unsigned int sector_number, unsigned int n_sectors,
                                      unsigned int sector_size)
{
  unsigned int x, i, a, c, n_words;

  /* convert the sector size into number of 4-byte words (it is assumed that sizeof(unsigned int) = 4) */

  n_words = sector_size / 4u;

  /* initialize the linear congruencial pseudo-random number generator
     (section 3.2.1.2 of The Art of Computer Programming presents the theory behind the restrictions on a and c) */

  i = sector_number;                                       // get the sector number
  a = 0xCCE00001u ^ ((i & 0x0F0F0F0Fu) << 2);              // a must be a multiple of 4 plus 1
  c = 0x00CCE001u ^ ((i & 0xF0F0F0F0u) >> 3);              // c must be odd
  x = 0xCCE02021u;                                         // initial state

  /* modify the sector data */

  for (i = 0u; i < n_words; i++)
  { x = a * x + c;                                         // update the pseudo-random generator state
    sector_data[i] ^= x;                                   // modify the sector data
  }
}

__global__ static void modify_sector_cuda_kernel (unsigned int * __restrict__ sector_data, unsigned int * __restrict__ sector_number,
                                           unsigned int n_sectors, unsigned int sector_size)
{
  unsigned int x, y, idx, i, a, c, n_words;

  /* compute the thread number */

  x = (unsigned int) threadIdx.x + (unsigned int) blockDim.x * (unsigned int) blockIdx.x;
  y = (unsigned int) threadIdx.y + (unsigned int) blockDim.y * (unsigned int) blockIdx.y;
  idx = (unsigned int) blockDim.x * (unsigned int) gridDim.x * y + x;
  if (idx >= n_sectors)
     return;                                             // safety precaution

  /* convert the sector size into number of 4-byte words (it is assumed that sizeof(unsigned int) = 4)
     and define boundaries */

  n_words = sector_size / 4u;

  /* adjust pointers */

  sector_data += n_words * idx;
  sector_number += idx;

  /* initialize the linear congruencial pseudo-random number generator
     (section 3.2.1.2 of The Art of Computer Programming presents the theory behind the restrictions on a and c) */

  i = sector_number[0];                                    // get the sector number
  a = 0xCCE00001u ^ ((i & 0x0F0F0F0Fu) << 2);              // a must be a multiple of 4 plus 1
  c = 0x00CCE001u ^ ((i & 0xF0F0F0F0u) >> 3);              // c must be odd
  x = 0xCCE02021u;                                         // initial state

  /* modify the sector data */

  for (i = 0u; i < n_words; i++)
  { x = a * x + c;                                          // update the pseudo-random generator state
    sector_data[i] ^= x;
  }
}

static double get_delta_time(void)
{
  static struct timespec t0,t1;

  t0 = t1;
  if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
  {
    perror("clock_gettime");
    exit(1);
  }
  return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}
